#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
* Copyright (c) 2013, Georgia Institute of Technology
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*
* 1. Redistributions of source code must retain the above copyright notice, this
* list of conditions and the following disclaimer.
* 2. Redistributions in binary form must reproduce the above copyright notice,
* this list of conditions and the following disclaimer in the documentation
* and/or other materials provided with the distribution.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
* AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
* SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
* CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
* OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/**********************************************
 * @file path_integral_main.cpp
 * @author Grady Williams <gradyrw@gmail.com>
 * @date May 24, 2017
 * @copyright 2017 Georgia Institute of Technology
 * @brief Main file model predictive path integral control.
 *
 ***********************************************/

//Some versions of boost require __CUDACC_VER__, which is no longer defined in CUDA 9. This is
//the old expression for how it was defined, so should work for CUDA 9 and under.
#define __CUDACC_VER__ __CUDACC_VER_MAJOR__ * 10000 + __CUDACC_VER_MINOR__ * 100 + __CUDACC_VER_BUILD__

// #include "../include/rcraicer_mppi/meta_math.h"
// #include "../include/rcraicer_mppi/param_getter.h"
// #include "../include/rcraicer_mppi/rcraicer_plant.h"
// // #include "../include/rcraicer_mppi/PathIntegralParamsConfig.h"
// #include "../include/rcraicer_mppi/path_integral/costs.cuh"

// //Including neural net model
// #ifdef MPPI_NNET_USING_CONSTANT_MEM__
// __device__ __constant__ float NNET_PARAMS[param_counter(6,32,32,4)];
// #endif
// #include "../include/rcraicer_mppi/neural_net_model.cuh"
// #include "../include/rcraicer_mppi/car_bfs.cuh"
// #include "../include/rcraicer_mppi/car_kinematics.cuh"
// #include "../include/rcraicer_mppi/generalized_linear.cuh"
// #include "../include/rcraicer_mppi/mppi_controller.cuh"
// #include "../include/rcraicer_mppi/run_control_loop.cuh"


#include "rclcpp/rclcpp.hpp"
// #include <atomic>

// #include <stdlib.h>
// #include <stdio.h>
// #include <math.h>

#include "../include/rcraicer_mppi/path_integral_node.cuh"

using namespace rcraicer_control;

using std::placeholders::_1;
using namespace std::chrono_literals;


PathIntegralNode::PathIntegralNode() : Node("path_integral_node"), robot(NULL)
{

  setupParams();
  updateParams();
  params.logger_name = "path_integral_node";
  params.use_sim_time = false;
  is_alive = true;

  paramSetCallbackHandler = this->add_on_set_parameters_callback(std::bind(&PathIntegralNode::paramSetCallback, this, std::placeholders::_1));  
  
  //Define the mppi costs  
  costs = new MPPICosts(config);

  //Define the internal dynamics model for mppi
  float2 control_constraints[2] = {make_float2(-.99, .99), make_float2(-.99, params.max_throttle)};
  model = new DynamicsModel(1.0/params.hz, control_constraints);
  model->loadParams(params.model_path, params.logger_name); //Load the model parameters from the launch file specified path  

  //Define the controller
  init_u[0] = (float)params.init_steering;
  init_u[1] = (float)params.init_throttle;

  exploration_std[0] = (float)params.steering_std;
  exploration_std[1] = (float)params.throttle_std;

  mppi = new Controller(model, costs, params.num_timesteps, params.hz, params.gamma, exploration_std, 
                                    init_u, params.num_iters, params.optimization_stride);

  robot = new RCRaicerPlant(params, config);  
  setupSubscribers();
  setupTimers();
  setupPublishers();
  robot->setPublishers(control_pub_, status_pub_, subscribed_pose_pub_, path_pub_, timing_data_pub_);

  // optimizer = std::thread(&runControlLoop<Controller>, mppi, robot, &params, &is_alive);
  optimizer = boost::thread(&runControlLoop<Controller>, mppi, robot, &params, &is_alive);

  RCLCPP_INFO(this->get_logger(), "Node started.");    
}

PathIntegralNode::~PathIntegralNode()
{
  pathTimer_->cancel();
  statusTimer_->cancel();
  debugImgTimer_->cancel();
  timingInfoTimer_->cancel();

  //Shutdown procedure
  is_alive.store(false);
  optimizer.join();
  robot->shutdown();
  mppi->deallocateCudaMem();
  delete robot;
  delete mppi;
  delete costs;
  delete model;
}


void PathIntegralNode::setupSubscribers()
{
  //Initialize the subscribers.
  pose_sub_ = this->create_subscription<nav_msgs::msg::Odometry>(params.pose_estimate_name, 1, std::bind(&RCRaicerPlant::poseCall, robot, std::placeholders::_1));
  sim_sub_ = this->create_subscription<rcraicer_msgs::msg::SimState>("sim_state", 1, std::bind(&RCRaicerPlant::simStateCall, robot, std::placeholders::_1));
  servo_sub_ = this->create_subscription<rcraicer_msgs::msg::ChassisState>("chassis_state", 1, std::bind(&RCRaicerPlant::servoCall, robot, std::placeholders::_1));
  model_sub_ = this->create_subscription<rcraicer_msgs::msg::NeuralNetModel>("/model_updater/model", 1, std::bind(&RCRaicerPlant::modelCall, robot, std::placeholders::_1));
}

void PathIntegralNode::setupPublishers()
{
    //Initialize the publishers.
    control_pub_ = this->create_publisher<rcraicer_msgs::msg::ChassisCommand>("chassis_cmds", 1);
    path_pub_ = this->create_publisher<nav_msgs::msg::Path>("nominalPath", 1);
    subscribed_pose_pub_ = this->create_publisher<nav_msgs::msg::Odometry>("subscribedPose", 1);
    status_pub_ = this->create_publisher<rcraicer_msgs::msg::PathIntegralStatus>("mppiStatus", 1);
    timing_data_pub_ = this->create_publisher<rcraicer_msgs::msg::PathIntegralTiming>("timingInfo", 1);
}

void PathIntegralNode::setupTimers()
{
    //Timer callback for path publisher
    pathTimer_ = this->create_wall_timer(std::chrono::milliseconds(33), std::bind(&RCRaicerPlant::pubPath, robot));
    statusTimer_ = this->create_wall_timer(std::chrono::milliseconds(33), std::bind(&RCRaicerPlant::pubStatus, robot));
    debugImgTimer_ = this->create_wall_timer(std::chrono::milliseconds(33), std::bind(&RCRaicerPlant::displayDebugImage, robot));
    timingInfoTimer_ = this->create_wall_timer(std::chrono::milliseconds(33), std::bind(&RCRaicerPlant::pubTimingData, robot));
}

void PathIntegralNode::setupParams()
{
  // <!--Pose topic configuration parameters -->
  this->declare_parameter<std::string>("pose_estimate", "/ground_truth/state");
  this->declare_parameter<std::string>("path_topic", "/mppi_controller/nominal_path");
  
  // <!--Debug/Sim mode parameter -->
  this->declare_parameter<bool>("debug_mode", true);
  
  // <!--Setup parameters-->
  this->declare_parameter<int>("hz", 50);
  this->declare_parameter<int>("num_timesteps", 100);
  this->declare_parameter<double>("x_pos", 0.0);
  this->declare_parameter<double>("y_pos", 0.0);
  this->declare_parameter<double>("heading", 2.35);

  this->declare_parameter<int>("optimization_stride", 1);
  this->declare_parameter<bool>("use_feedback_gains", true);

  // Model parameters
  this->declare_parameter<std::string>("model_path", "");

  // <!-- Control hyper-parameters-->
  this->declare_parameter<double>("gamma", 0.15);
  this->declare_parameter<int>("num_iters", 1);

  // <!-- Control parameters -->
  this->declare_parameter<double>("init_steering", 0.0);
  this->declare_parameter<double>("init_throttle", 0.0);
  this->declare_parameter<double>("steering_std", 0.275);
  this->declare_parameter<double>("throttle_std", 0.3);
  this->declare_parameter<double>("max_throttle", 0.65);

  // <!-- Cost Parameters -->
  this->declare_parameter<bool>("l1_cost", false);
  this->declare_parameter<double>("desired_speed", 6.0);
  this->declare_parameter<double>("speed_coefficient", 4.25);
  this->declare_parameter<double>("track_coefficient", 200.0);
  this->declare_parameter<double>("max_slip_angle", 1.25);
  this->declare_parameter<double>("slip_penalty", 10.0);
  this->declare_parameter<double>("track_slop", 0.0);
  this->declare_parameter<double>("crash_coeff", 10000.0);
  this->declare_parameter<double>("steering_coeff", 0.0);
  this->declare_parameter<double>("throttle_coeff", 0.0);
  this->declare_parameter<double>("boundary_threshold", 0.65);
  this->declare_parameter<double>("discount", 0.1);
  this->declare_parameter<std::string>("map_path", "");
}

void PathIntegralNode::updateParams()
{      
    params.pose_estimate_name = this->get_parameter("pose_estimate").as_string();
    params.path_topic = this->get_parameter("path_topic").as_string();
    params.debug_mode = this->get_parameter("debug_mode").as_bool();
    params.hz = this->get_parameter("hz").as_int();
    params.num_timesteps = this->get_parameter("num_timesteps").as_int();
    params.num_iters = this->get_parameter("num_iters").as_int();
    params.x_pos = this->get_parameter("x_pos").as_double();
    params.y_pos = this->get_parameter("y_pos").as_double();
    params.heading = this->get_parameter("heading").as_double();
    params.gamma = this->get_parameter("gamma").as_double();
    params.init_steering = this->get_parameter("init_steering").as_double();
    params.init_throttle = this->get_parameter("init_throttle").as_double();
    params.steering_std = this->get_parameter("steering_std").as_double();
    params.throttle_std = this->get_parameter("throttle_std").as_double();
    params.max_throttle = this->get_parameter("max_throttle").as_double();
    params.model_path = this->get_parameter("model_path").as_string();    
    params.optimization_stride = this->get_parameter("optimization_stride").as_int();
    params.use_feedback_gains = this->get_parameter("use_feedback_gains").as_bool();


    if (config.desired_speed != this->get_parameter("desired_speed").as_double())
      costParamsUpdated = true;

    if (config.speed_coefficient != this->get_parameter("speed_coefficient").as_double())
        costParamsUpdated = true;

    if (config.track_coefficient != this->get_parameter("track_coefficient").as_double())
      costParamsUpdated = true;

    if (config.max_slip_angle != this->get_parameter("max_slip_angle").as_double())
      costParamsUpdated = true;

    if (config.slip_penalty != this->get_parameter("slip_penalty").as_double())
      costParamsUpdated = true;

    if (config.crash_coefficient != this->get_parameter("crash_coeff").as_double())
      costParamsUpdated = true;

    if (config.track_slop != this->get_parameter("track_slop").as_double())
      costParamsUpdated = true;

    if (config.steering_coeff != this->get_parameter("steering_coeff").as_double())
      costParamsUpdated = true;

    if (config.throttle_coeff != this->get_parameter("throttle_coeff").as_double())
      costParamsUpdated = true;
      
    config.max_throttle = this->get_parameter("max_throttle").as_double();
    config.desired_speed = this->get_parameter("desired_speed").as_double();
    config.speed_coefficient = this->get_parameter("speed_coefficient").as_double();  
    config.track_coefficient = this->get_parameter("track_coefficient").as_double();  
    config.max_slip_angle = this->get_parameter("max_slip_angle").as_double();  
    config.slip_penalty = this->get_parameter("slip_penalty").as_double();  
    config.crash_coefficient = this->get_parameter("crash_coeff").as_double();  
    config.track_slop = this->get_parameter("track_slop").as_double();  
    config.steering_coeff = this->get_parameter("steering_coeff").as_double();  
    config.throttle_coeff = this->get_parameter("throttle_coeff").as_double();  
    config.boundary_threshold = this->get_parameter("boundary_threshold").as_double();  
    config.discount = this->get_parameter("discount").as_double();  
    config.num_timesteps = this->get_parameter("num_timesteps").as_int();  
    config.l1_cost = this->get_parameter("l1_cost").as_bool();  
    config.map_path = this->get_parameter("map_path").as_string();    

    if (costParamsUpdated && robot != NULL)
    {      
      robot->updateConfigParams(config);
    }

    costParamsUpdated = false;
}

rcl_interfaces::msg::SetParametersResult PathIntegralNode::paramSetCallback(const std::vector<rclcpp::Parameter>& parameters)
{
    rcl_interfaces::msg::SetParametersResult result;
    result.successful = true;

    updateParams();
    
    return result;
}

int main(int argc, char** argv) {  
  //Ros node initialization
  rclcpp::init(argc, argv);
  rclcpp::spin(std::make_shared<PathIntegralNode>());  
  rclcpp::shutdown();
  return 0;   
}
