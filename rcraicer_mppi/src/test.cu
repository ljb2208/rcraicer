#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
* Copyright (c) 2013, Georgia Institute of Technology
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*
* 1. Redistributions of source code must retain the above copyright notice, this
* list of conditions and the following disclaimer.
* 2. Redistributions in binary form must reproduce the above copyright notice,
* this list of conditions and the following disclaimer in the documentation
* and/or other materials provided with the distribution.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
* AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
* SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
* CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
* OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/**********************************************
 * @file path_integral_main.cpp
 * @author Grady Williams <gradyrw@gmail.com>
 * @date May 24, 2017
 * @copyright 2017 Georgia Institute of Technology
 * @brief Main file model predictive path integral control.
 *
 ***********************************************/

//Some versions of boost require __CUDACC_VER__, which is no longer defined in CUDA 9. This is
//the old expression for how it was defined, so should work for CUDA 9 and under.
#define __CUDACC_VER__ __CUDACC_VER_MAJOR__ * 10000 + __CUDACC_VER_MINOR__ * 100 + __CUDACC_VER_BUILD__

#include "../include/rcraicer_mppi/meta_math.h"


//Including neural net model
#ifdef MPPI_NNET_USING_CONSTANT_MEM__
__device__ __constant__ float NNET_PARAMS[param_counter(6,32,32,4)];
#endif


#include "../include/rcraicer_mppi/neural_net_model.cuh"
#include "../include/rcraicer_mppi/car_bfs.cuh"
#include "../include/rcraicer_mppi/car_kinematics.cuh"
#include "../include/rcraicer_mppi/generalized_linear.cuh"


#include <atomic>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <chrono>
#include <functional>
#include <cfloat>

using namespace rcraicer_control;

const int MPPI_NUM_ROLLOUTS__ = 1920;
const int BLOCKSIZE_X = 8;
const int BLOCKSIZE_Y = 16;
typedef NeuralNetModel<7,2,3,6,32,32,4> DynamicsModel;

int main(int argc, char** argv) {  
  //Define the internal dynamics model for mppi
  float2 control_constraints[2] = {make_float2(-.99, .99), make_float2(-.99, .99)};
  DynamicsModel* model = new DynamicsModel(1.0/50.0, control_constraints);
  model->loadParams("/home/lbarnett/catkin_ws/src/autorally/autorally_control/src/path_integral/params/models/autorally_nnet_09_12_2018.npz", ""); //Load the model parameters from the launch file specified path  

  DynamicsModel* model2 = new DynamicsModel(1.0/50.0, control_constraints);
  model2->loadParams("/home/lbarnett/ros2_ws/src/rcraicer/rcraicer_mppi_test/models/dynmodel_states4.npz", ""); //Load the model parameters from the launch file specified path  

  Eigen::MatrixXf s(7,1);
  s(0) = 0.0;
  s(1) = 0.0;
  s(2) = 0.0;
  s(3) = 0.0;
  s(4) = 0.0;
  s(5) = 0.0;
  s(6) = 0.0;
  Eigen::MatrixXf u(2,1);
  u(0) = 0.0;
  u(1) = 0.5;

  model->computeDynamics(s, u);

  std::cout << "Model1: ";
  for (int i=3; i < 7; i++)
  {
    std::cout << model->state_der_(i) << " ";
  }

  std::cout << "\nModel2: ";

  model2->computeDynamics(s, u);

  for (int i=3; i < 7; i++)
  {
    std::cout << model2->state_der_(i) << " ";
  }


  delete model;
  delete model2;
  return 0;   
}
